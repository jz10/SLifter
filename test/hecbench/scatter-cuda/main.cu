#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include "TensorInfo.h"
#include "reducer.h"

#define THREADS 256
#define BLOCKS(N) (N + THREADS - 1) / THREADS

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = func;                                                 \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
    }                                                                          \
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const scalar_t *src_data,
               const TensorInfo<int64_t, int64_t> index_info,
               scalar_t *out_data, int E, int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = IndexToOffset<int64_t, int64_t, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    Reducer<scalar_t, REDUCE>::atomic_write(out_data + b * N * K + idx * K + k,
                                            src_data[thread_idx]);
  }
}

template<typename scalar_t, ReductionType REDUCE>
void scatter(int64_t num_elems, int repeat) {
  int64_t out_size = num_elems / 2;
  int64_t src_size_bytes = num_elems * sizeof(scalar_t);
  int64_t idx_size_bytes = num_elems * sizeof(int64_t);
  int64_t out_size_bytes = out_size * sizeof(scalar_t);

  int64_t *h_idx = (int64_t*) malloc (idx_size_bytes);
  scalar_t *h_src = (scalar_t*) malloc (src_size_bytes);
  scalar_t *h_out = (scalar_t*) malloc (out_size_bytes);
  scalar_t *h_ref = (scalar_t*) malloc (out_size_bytes);
  srand(123);
  for (int64_t i = 0; i < num_elems; i++) {
    h_idx[i] = rand() % out_size;
    h_src[i] = rand() % 2 ? 1 : -1;
  }

  scalar_t *d_src, *d_out;
  int64_t *d_idx;
  CHECK_CUDA( hipMalloc(&d_src, src_size_bytes) )
  CHECK_CUDA( hipMalloc(&d_out, out_size_bytes) )
  CHECK_CUDA( hipMalloc(&d_idx, idx_size_bytes) )
  CHECK_CUDA( hipMemcpy(d_idx, h_idx, idx_size_bytes, hipMemcpyHostToDevice) )
  CHECK_CUDA( hipMemcpy(d_src, h_src, src_size_bytes, hipMemcpyHostToDevice) )

  int64_t tensor_sizes[] = {num_elems};
  int64_t tensor_strides[] = {1};

  TensorInfo<int64_t, int64_t> index_info(d_idx, 1, tensor_sizes, tensor_strides);

  dim3 grids (BLOCKS(num_elems));
  dim3 blocks (THREADS);

  int B = 1;
  int E = num_elems;
  int K = num_elems / (B * E);
  int N = num_elems / 2;

  for (int64_t i = 0; i < out_size; i++) {
    h_ref[i] = h_out[i] = Reducer<scalar_t, REDUCE>::init();
  }
  
  int64_t time = 0;
  for (int i = 0; i < repeat; i++) {
    CHECK_CUDA( hipMemcpy(d_out, h_out, out_size_bytes, hipMemcpyHostToDevice) )
    CHECK_CUDA( hipDeviceSynchronize() )
    auto start = std::chrono::steady_clock::now();
    scatter_kernel<scalar_t, REDUCE> <<<grids, blocks>>>
        (d_src, index_info, d_out, E, K, N, num_elems);
    CHECK_CUDA( hipDeviceSynchronize() )
    auto end = std::chrono::steady_clock::now();
    time += std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  }
  printf("Average execution time of kernel: %f (us)\n", (time * 1e-3f) / repeat);

  CHECK_CUDA( hipMemcpy(h_out, d_out, out_size_bytes, hipMemcpyDeviceToHost) )

  for (int64_t i = 0; i < num_elems; i++) {
    Reducer<scalar_t, REDUCE>::update(h_ref[h_idx[i]], h_src[i]);
  }

  bool ok = true;
  for (int64_t i = 0; i < out_size; i++) {
    if ((double)h_out[i] - (double)h_ref[i] > 1e-3) {
      ok = false;
      break;
    }
  }
  printf("%s\n\n", ok ? "PASS" : "FAIL");

  CHECK_CUDA( hipFree(d_src) )
  CHECK_CUDA( hipFree(d_idx) )
  CHECK_CUDA( hipFree(d_out) )
  free(h_src);
  free(h_idx);
  free(h_out);
  free(h_ref);
}

int main(int argc, char* argv[])
{
  if (argc != 3) {
    printf("Usage: %s <number of elements> <repeat>\n", argv[0]);
    return 1;
  }
  const int64_t num_elements = atol(argv[1]);
  const int repeat = atoi(argv[2]);
  printf("INT32 scatter (mul, div, sum, min, max)\n"); 
  scatter<int32_t, MUL>(num_elements, repeat);
  scatter<int32_t, DIV>(num_elements, repeat);
  scatter<int32_t, SUM>(num_elements, repeat);
  scatter<int32_t, MIN>(num_elements, repeat);
  scatter<int32_t, MAX>(num_elements, repeat);
  printf("INT64 scatter (mul, div, sum, min, max)\n"); 
  scatter<int64_t, MUL>(num_elements, repeat);
  scatter<int64_t, DIV>(num_elements, repeat);
  scatter<int64_t, SUM>(num_elements, repeat);
  scatter<int64_t, MIN>(num_elements, repeat);
  scatter<int64_t, MAX>(num_elements, repeat);
  printf("FP32 scatter (mul, div, sum, min, max)\n"); 
  scatter<float, MUL>(num_elements, repeat);
  scatter<float, DIV>(num_elements, repeat);
  scatter<float, SUM>(num_elements, repeat);
  scatter<float, MIN>(num_elements, repeat);
  scatter<float, MAX>(num_elements, repeat);
  printf("FP64 scatter (mul, div, sum, min, max)\n"); 
  scatter<double, MUL>(num_elements, repeat);
  scatter<double, DIV>(num_elements, repeat);
  scatter<double, SUM>(num_elements, repeat);
  scatter<double, MIN>(num_elements, repeat);
  scatter<double, MAX>(num_elements, repeat);
  return 0;
}
