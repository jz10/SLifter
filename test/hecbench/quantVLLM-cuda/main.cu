#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cub/util_type.cuh>
#include <hipcub/hipcub.hpp>
#include "utils.h"
#include "reference.h"

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const scale, const int hidden_size) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;

  // Performed using 64-bit math to avoid integer overflow.
  out += token_idx * hidden_size;
  input += token_idx * hidden_size;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[i] = float_to_int8_rn(static_cast<float>(input[i]) / scale);
  }
}

template <typename scalar_t, typename scale_type, typename azp_type>
__global__ void static_scaled_int8_azp_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const scale, azp_type const azp,
    const int hidden_size) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;

  // Performed using 64-bit math to avoid integer overflow.
  out += token_idx * hidden_size;
  input += token_idx * hidden_size;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    auto const val = static_cast<float>(input[i]);
    auto const quant_val = int32_to_int8(float_to_int32_rn(val / scale) + azp);
    out[i] = quant_val;
  }
}

template <typename scalar_t, typename scale_type>
__global__ void dynamic_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size) {
  int const tid = threadIdx.x;
  int64_t const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  float const zero = 0.0f;

  // Performed using 64-bit math to avoid integer overflow.
  out += token_idx * hidden_size;
  input += token_idx * hidden_size;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = static_cast<float>(input[i]);
    val = val > zero ? val : -val;
    absmax_val = val > absmax_val ? val : absmax_val;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  float const block_absmax_val_maybe =
      BlockReduce(reduceStorage).Reduce(absmax_val, hipcub::Max{}, blockDim.x);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[i] = float_to_int8_rn(static_cast<float>(input[i]) * tmp_scale);
  }
}

template <typename scalar_t, typename scale_type, typename azp_type>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, azp_type* azp, const int hidden_size) {
  int64_t const token_idx = blockIdx.x;

  // Performed using 64-bit math to avoid integer overflow.
  out += token_idx * hidden_size;
  input += token_idx * hidden_size;

  // Scan for the min and max value for this token
  float max_val = std::numeric_limits<float>::min();
  float min_val = std::numeric_limits<float>::max();
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto val = static_cast<float>(input[i]);
    max_val = std::max(max_val, val);
    min_val = std::min(min_val, val);
  }

  // Reduce the max and min values across the block
  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStorage;
  max_val = BlockReduce(reduceStorage).Reduce(max_val, hipcub::Max{}, blockDim.x);
  __syncthreads();  // Make sure min doesn't mess with max shared memory
  min_val = BlockReduce(reduceStorage).Reduce(min_val, hipcub::Min{}, blockDim.x);

  __shared__ scale_type scale_sh;
  __shared__ azp_type azp_sh;

  // Compute the scale and zero point and store them, only on the first thread
  if (threadIdx.x == 0) {
    float const scale_val = (max_val - min_val) / 255.0f;
    // Use rounding to even (same as torch.round)
    auto const azp_float = std::nearbyint(-128.0f - min_val / scale_val);
    auto const azp_val = static_cast<azp_type>(azp_float);

    // Store the scale and azp into shared and global
    scale[token_idx] = scale_sh = scale_val;
    azp[token_idx] = azp_sh = azp_val;
  }

  // Wait for the scale and azp to be computed
  __syncthreads();

  float const scale_val = scale_sh;
  azp_type const azp_val = azp_sh;

  // Quantize the values
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto const val = static_cast<float>(input[i]);
    auto const quant_val =
        int32_to_int8(float_to_int32_rn(val / scale_val) + azp_val);
    out[i] = quant_val;
  }
}


template <typename scalar_t>
void static_scaled_int8_quant(int8_t *output,
                              const scalar_t *input,
                              int const hidden_size,
                              int const num_tokens,
                              float scale,
                              int32_t azp)
{
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));

  if (!azp)
    static_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block>>>(input, output, scale, hidden_size);
  else
    static_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block>>>(input, output, scale, azp, hidden_size);
}

template <typename scalar_t>
void dynamic_scaled_int8_quant(int8_t *output,
                               const scalar_t *input,
                               int const hidden_size,
                               int const num_tokens,
                               float *scales,
                               int32_t *azp)
{
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  if (azp == nullptr) {
    dynamic_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block>>>(input, output, scales, hidden_size);
  } else {
    dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block>>>(input, output, scales, azp, hidden_size);
  }
}

template <typename scalar_t>
void quant(int num_tokens, int hidden_size, int repeat) {
  
  scalar_t *d_input, *h_input;
  float *d_scale, *h_scale;
  int32_t *d_azp = nullptr, *h_azp;
  int8_t *d_output, *h_output, *h_output_r;


  size_t input_size_bytes = hidden_size * num_tokens * sizeof(scalar_t);
  size_t output_size_bytes = hidden_size * num_tokens * sizeof(int8_t);
  size_t scale_size_bytes = num_tokens * sizeof(float);
  size_t azp_size_bytes = num_tokens * sizeof(int32_t);

  srand(123);
  h_input = (scalar_t*) malloc (input_size_bytes);
  for (int i = 0; i < hidden_size * num_tokens; i++) {
    h_input[i] = rand() % 1000 - 300;
  }
  
  h_scale = (float*) malloc (scale_size_bytes);
  h_azp = (int32_t*) malloc (azp_size_bytes);
  h_output = (int8_t*) malloc (output_size_bytes);
  h_output_r = (int8_t*) malloc (output_size_bytes);

  hipMalloc(&d_input, input_size_bytes);
  hipMalloc(&d_output, output_size_bytes);
  hipMalloc(&d_scale, scale_size_bytes);

  hipMemcpy(d_input, h_input, input_size_bytes, hipMemcpyHostToDevice);

  float scale = 0.1;
  int32_t error = 0;

  // static_scaled_int8_quant
  int32_t azp = 0;
  auto start = std::chrono::steady_clock::now();
  for (int i = 0; i < repeat; i++) 
    static_scaled_int8_quant(d_output, d_input, hidden_size, num_tokens, scale, azp);
  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of static_scaled_int8_quant kernel: %f (us)\n",
          (time * 1e-3f) / repeat);
  hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);
  static_scaled_int8_quant_reference(h_input, h_output_r, scale, num_tokens, hidden_size);
  error = memcmp(h_output, h_output_r, output_size_bytes);

  // static_scaled_int8_quant_azp
  azp = 54;
  start = std::chrono::steady_clock::now();
  for (int i = 0; i < repeat; i++) 
    static_scaled_int8_quant(d_output, d_input, hidden_size, num_tokens, scale, azp);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of static_scaled_int8_quant_azp kernel: %f (us)\n",
          (time * 1e-3f) / repeat);
  hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);
  static_scaled_int8_azp_quant_reference(h_input, h_output_r, scale, azp, num_tokens, hidden_size);
  error += memcmp(h_output, h_output_r, output_size_bytes);

  // dynamic_scaled_int8_quant
  start = std::chrono::steady_clock::now();
  for (int i = 0; i < repeat; i++) 
    dynamic_scaled_int8_quant(d_output, d_input, hidden_size, num_tokens, d_scale, d_azp);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of dynamic_scaled_int8_quant kernel: %f (us)\n",
          (time * 1e-3f) / repeat);
  hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);
  dynamic_scaled_int8_quant_reference(h_input, h_output_r, h_scale, num_tokens, hidden_size);
  error += memcmp(h_output, h_output_r, output_size_bytes);

  // dynamic_scaled_int8_quant_azp
  hipMalloc(&d_azp, azp_size_bytes);
  start = std::chrono::steady_clock::now();
  for (int i = 0; i < repeat; i++) 
    dynamic_scaled_int8_quant(d_output, d_input, hidden_size, num_tokens, d_scale, d_azp);
  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of dynamic_scaled_int8_quant_azp kernel: %f (us)\n",
          (time * 1e-3f) / repeat);
  hipMemcpy(h_output, d_output, output_size_bytes, hipMemcpyDeviceToHost);
  dynamic_scaled_int8_azp_quant_reference(h_input, h_output_r, h_scale, h_azp, num_tokens, hidden_size);
  error += memcmp(h_output, h_output_r, output_size_bytes);

  printf("%s\n", error ? "FAIL" : "PASS");

  hipFree(d_input);
  hipFree(d_output);
  hipFree(d_scale);
  hipFree(d_azp);
  free(h_input);
  free(h_output);
  free(h_output_r);
  free(h_scale);
  free(h_azp);
}

int main(int argc, char* argv[])
{
  if (argc != 4) {
    printf("Usage: %s <number of tokens> <hidden size> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_tokens = atoi(argv[1]);
  const int hidden_size = atoi(argv[2]);
  const int repeat = atoi(argv[3]);

  printf("Input type is FP16\n");
  quant<__half>(num_tokens, hidden_size, repeat);

  printf("Input type is BF16\n");
  quant<__hip_bfloat16>(num_tokens, hidden_size, repeat);

  printf("Input type is FP32\n");
  quant<float>(num_tokens, hidden_size, repeat);
}
