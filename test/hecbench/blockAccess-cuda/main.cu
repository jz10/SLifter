#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <hip/hip_runtime.h>
#include "block_load.h"
#include "block_store.h"

#define NUM 4

__global__ void reference (const float * __restrict__ A,
                           unsigned char *out, const unsigned int n)
{
  for (unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
       idx < n/4; idx += gridDim.x * blockDim.x) {
    const float4 v = reinterpret_cast<const float4*>(A)[idx];
    uchar4 o;
    o.x = (int)v.x;
    o.y = (int)v.y;
    o.z = (int)v.z;
    o.w = (int)v.w;
    reinterpret_cast<uchar4*>(out)[idx] = o;
  }
}

template<int TH, int ITEMS_TO_LOAD>
__global__ void kernel (const float * __restrict__ A,
                        unsigned char *out, const unsigned int n)
{
  const int bid = blockIdx.x;
  const int base_idx = (bid * ITEMS_TO_LOAD);

  float vals[NUM];
  unsigned char qvals[NUM];

  // 1D block of TH threads owning NUM integer items each
  typedef BlockLoad<float, TH, NUM> LoadFloat;
  typedef BlockStore<unsigned char, TH, NUM> StoreChar;

  __shared__ typename LoadFloat::TempStorage loadf_storage;
  __shared__ typename StoreChar::TempStorage storec_storage;

  for (int i = base_idx; i < n; i += gridDim.x*ITEMS_TO_LOAD)
  {
      int valid_items = min(n - i, ITEMS_TO_LOAD);

      // Parameters:
      // block_src_it – [in] The thread block's base iterator for loading from
      // dst_items – [out] Destination to load data into
      // block_items_end – [in] Number of valid items to load
      LoadFloat(loadf_storage).Load(&(A[i]), vals, valid_items);

      #pragma unroll
      for(int j = 0; j < NUM; j++)
          qvals[j] = int(vals[j]);

      StoreChar(storec_storage).Store(&(out[i]), qvals, valid_items);
  }
}

int main(int argc, char* argv[])
{
  if (argc != 4) {
    printf("Usage: %s <number of rows> <number of columns> <repeat>\n", argv[0]);
    return 1;
  }
  const int nrows = atoi(argv[1]);
  const int ncols = atoi(argv[2]);
  const int repeat = atoi(argv[3]);

  const unsigned int n = nrows * ncols;
  const size_t A_size = n * sizeof(float);
  const size_t out_size = n * sizeof(unsigned char);

  float *A = (float*) malloc (A_size);
  unsigned char *out = (unsigned char*) malloc (out_size);

  std::mt19937 gen{19937};
 
  std::normal_distribution<float> d{128.0, 127.0};

  for (unsigned int i = 0; i < n; i++) {
    A[i] = d(gen); 
  }

  float *d_A;
  hipMalloc((void**)&d_A, A_size);
  hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice);

  unsigned char *d_out;
  hipMalloc((void**)&d_out, out_size);
  
  const int block_size = 256;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  dim3 grid (16 * prop.multiProcessorCount);
  dim3 block (block_size);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    reference<<<grid, block>>>(d_A, d_out, n);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of the reference kernel: %f (us)\n", (time * 1e-3f) / repeat);

  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    kernel<block_size, block_size*NUM><<<grid, block>>>(d_A, d_out, n);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of the blockAccess kernel: %f (us)\n", (time * 1e-3f) / repeat);
  
  hipMemcpy(out, d_out, out_size, hipMemcpyDeviceToHost);

  bool error = false;
  for (unsigned int i = 0; i < n; i++) {
    unsigned char t = int(A[i]);
    if (out[i] != t) {
      printf("@%u: %u != %u\n", i, out[i], t);
      error = true;
      break;
    }
  }
  printf("%s\n", error ? "FAIL" : "PASS");
  
  hipFree(d_A);
  hipFree(d_out);
  free(A);
  free(out);
  return 0;
}
