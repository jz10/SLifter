#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void loop1(const float* A, const float* B, float* C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    int sum = 0;
    #pragma unroll 1
    for (int i = idx; i < n; i += 1)
        sum += i;

    C[idx] = sum;
}

int main() {
    int n = 1 << 20;
    size_t sz = n * sizeof(float);
    float *h_A = (float*)malloc(sz), *h_B = (float*)malloc(sz), *h_C = (float*)malloc(sz);
    for (int i = 0; i < n; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, sz);
    hipMalloc(&d_B, sz);
    hipMalloc(&d_C, sz);
    hipMemcpy(d_A, h_A, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sz, hipMemcpyHostToDevice);
    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    loop1<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipMemcpy(h_C, d_C, sz, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; i++)
        printf("%f ", h_C[i]);
    printf("\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
